#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

#include "utils.h"
#include "scene.h"

#define pi 3.14159265358979323

__device__ Vec3 RayTrace(const World* world, const Vec3& rayOrigin, const Vec3& rayDireciton);

struct raytraceInfo {

    int IMG_HEIGHT;
    int IMG_WIDTH;

    Quat cameraRot;
    Vec3 cameraPos;
    float focal_length;

};

struct Int3 {

    int x;
    int y;
    int z;

    CUDA_HEAD Int3()
    : x(0), y(0), z(0)
    {}

    CUDA_HEAD Int3(int a, int b, int c)
    : x(a), y(b), z(c) 
    {}

    CUDA_HEAD Int3(Vec3 in)
    : Int3(
        static_cast<int>(in.x), 
        static_cast<int>(in.y),
        static_cast<int>(in.z)) 
    {}

    
};

__global__ void rayTraceGPU(Int3* colors, const World* world, const raytraceInfo& info) {

    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    
    int index = row * info.IMG_WIDTH + col;

    colors[index] = Int3(255, 255, 255);
    return;

    if (row > info.IMG_HEIGHT || col > info.IMG_WIDTH)
        return;

    
    float y = float(row), z = float(info.IMG_HEIGHT - col - 1);

    float alphay = y / (float) (info.IMG_WIDTH - 1);
    float alphaz = z / (float) (info.IMG_HEIGHT - 1);

    Vec3 rel_viewpoint_pos = Vec3(info.focal_length, -0.5 + alphaz, 0.5 - alphay);
    rel_viewpoint_pos = info.cameraRot.RotateVector(rel_viewpoint_pos);

    Vec3 color = RayTrace(world, info.cameraPos, rel_viewpoint_pos.normalize());

    colors[index] = Int3(color * 255.999);
    colors[index] = Int3(255, 255, 255);
}

__device__ Vec3 RayTrace(const World* world, const Vec3& rayOrigin, const Vec3& rayDireciton) {

    RayIntersectInfo closestRay{};

    // First run-through to determine what object it hits
    // for (SceneObject* object : world->objects) {
    for (int i = 0; i < MAX_OBJS; ++i) {
        const SceneObject* object = world->objects[i];

        RayIntersectInfo rayInfo{};

        if (object->RayObjectIntersect(rayOrigin, rayDireciton, rayInfo) 
            && rayInfo.distance < closestRay.distance)
            closestRay = rayInfo;

    }

    // No hit; return background color
    if (closestRay.distance == INFINITY)
        return Vec3(0.3, 0.4, 0.7);

    float lightIntensity = 0.0;
    float maxIntensity = 0.0;
    // for (LightSource* light : world->lights) {
    for (int i = 0; i < MAX_LGHT; ++i) {

        const LightSource* light = world->lights[i];

        maxIntensity += light->intensity;

        Vec3 lightDir = light->getDirection(closestRay.intersectPoint);

        RayIntersectInfo shadowRayInfo{};

        // Second run-through for shadows (and maybe reflections too idk)
        // for (SceneObject* object : world->objects) {
        for (int i = 0; i < MAX_OBJS; ++i) {
            const SceneObject* object = world->objects[i];

            if (object == closestRay.hitObj || object->castShadow == false)
                continue; 

            RayIntersectInfo rayInfo{};

            if (object->RayObjectIntersect(closestRay.intersectPoint, -1.0 * lightDir, rayInfo, 0.0001)
                && rayInfo.distance < shadowRayInfo.distance)
                    shadowRayInfo = rayInfo;

        }

        if (shadowRayInfo.distance == INFINITY) { // No hit
            lightIntensity += light->calculateIntensity(closestRay.planeNormal, 
                closestRay.intersectPoint);
        }
        
    }

    lightIntensity /= maxIntensity;
    
    // Vec3 P = rayOrigin + closest_dist * rayDireciton;
    // Vec3 SphereNormal = closest_object.NormalAtPoint(P); // (P - closest_object.position) / closest_object.radius;
    // float alpha = (1.0 - world->lightDirection.dot(closestIntersect.planeNormal)) * 0.5;
    return lightIntensity * closestRay.pointColor;
}

int main(int argc, char** argv) {

    const int IMG_HEIGHT = 1600;
    const int IMG_WIDTH = 1600;

    World* w = new World();
    Sphere* s;
    
    // s = new Sphere(Vec3(5.0, 0.0, 2.0), 2.0);
    // s->color = Vec3(0.0, 1.0, 0.0);
    // w->objects.push_back(s);

    s = new Sphere(Vec3(0.0, 0.0, 1.5), 1.0);
    s->color = Vec3(0.0, 0.0, 1.0);
    // w->objects.push_back(s);
    w->objects[0] = s;

    Sphere* gnd = new Sphere(Vec3(0.0, 0.0, -500.0), 500.0);
    gnd->color = Vec3(0.5, 0.0, 0.5).normalize();
    // w->objects.push_back(gnd);
    w->objects[1] = gnd;

    // Where the light is coming from
    Vec3 lightDir = Vec3(0.0, 0.0, -1.0).normalize();
    // w->lights.push_back(new DirectionalLight(lightDir, 1.0));
    w->lights[0] = new DirectionalLight(lightDir, 1.0);

    PointLight* p = new PointLight();
    p->intensity = 0.5;
    p->position = Vec3(1.0, 1.0, 3.0);
    // w->lights.push_back(p);
    w->lights[1] = p;

    s = new Sphere(p->position, 0.5);
    s->color = Vec3(1.0);
    s->castShadow = false;
    // w->objects.push_back(s);
    w->objects[2] = s;

    // SceneObject lightSource = SceneObject();


    // TriMesh* t = new TriMesh("../cube.obj");
    // // TriMesh* t = new TriMesh();
    
    // t->position = Vec3(3.0, 0.0, 0.0);
    // // t->scale = Vec3(0.1, 1.0, 5.0);

    // // t->verts.push_back(Vec3(0.0, 0.0, 0.0));
    // // t->verts.push_back(Vec3(0.0, 3.0, 0.0));
    // // t->verts.push_back(Vec3(0.0, 0.0, 3.0));
    // // t->verts.push_back(Vec3(0.0, 3.0, 3.0));

    // t->color = Vec3(1.0, 1.0, 0.0);

    // // t->tris.push_back(TriangleInfo(2, 1, 0));
    // // t->tris.push_back(TriangleInfo(2, 3, 1));

    // w->objects.push_back(t);

    // for (Vec3 v : t->verts) {
    //     w->objects.push_back(new Sphere(v + t->position, 0.25));
    // }

    char filename[32];
    const float focal_length = 0.5;

    int N_FRAMES;
    if (argc == 2)
        N_FRAMES = atoi(argv[1]);
    else
        N_FRAMES = 1;

    const Quat start_rot = Quat(Vec3::up(), 0.0 * pi / 180.0);
    const Quat end_rot = Quat(Vec3::up(), 180.0 * pi / 180.0); // * Quat(Vec3::forward(), 3.14159);
    
    Quat lilRot = Quat(Vec3::up(), 8.0 * (pi / (float) N_FRAMES));

    // const Vec3 start_loc = Vec3(-1.0, 3.0, 1.0);
    // const Vec3 middle_loc = Vec3(-4.0, 0.0, 5.5);
    // const Vec3 end_loc = Vec3(-1.0, 3.0, 11.0);

    size_t size_data = IMG_WIDTH * IMG_HEIGHT * sizeof(Int3);
    Int3* gpuColors;
    Int3* colors = new Int3[IMG_WIDTH * IMG_HEIGHT];

    hipMalloc((void**) &gpuColors, size_data);

    int threadsPer = 16;
    // block dimension
    dim3 Block(threadsPer, threadsPer);
    // grid of blocks
    // dim3 Grid(IMG_WIDTH / threadsPer, IMG_HEIGHT / threadsPer);
    dim3 Grid(1, 1);

    raytraceInfo rtInfo;

    rtInfo.IMG_HEIGHT = IMG_HEIGHT;
    rtInfo.IMG_WIDTH = IMG_WIDTH;
    rtInfo.focal_length = focal_length;
    
    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms; // which is applicable for asynchronous code also
    hipError_t errorcode;

    int gpucount = 0;

    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice) {
        printf("No GPUs are visible\n");
        exit(-1);
    }
    else {
        printf("Device count = %d\n",gpucount);
    }
    
    hipSetDevice(0);
    printf("Using device %d\n",0);

    for (int frame = 0; frame < N_FRAMES; ++frame) {

        float frameAlpha = 0.0;

        if (N_FRAMES > 1)
            frameAlpha = float(frame) / float(N_FRAMES - 1);

        // Create the image file for this frame
        sprintf(filename, "./images/image%04d.ppm", frame);

        std::ofstream imgout(filename); // std::format("./images/image{frame:4d}.ppm"));

        Quat rot = Quat(Vec3::up(), pi * 2.0 * frameAlpha); 
        Vec3 cameraPos = rot.RotateVector(Vec3(-5.0, 0.0, 1.0));

        imgout << "P3\n" << IMG_WIDTH << ' ' << IMG_HEIGHT << "\n255\n";


        rtInfo.cameraPos = cameraPos;
        rtInfo.cameraRot = rot;

        // Timing stuff start!
        hipEventCreate(&start); // instrument code to measure start time
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        // Calls the kernel
        printf("Launching kernel\n");
        rayTraceGPU<<<Grid, Block>>>(gpuColors, w, rtInfo);

        // Timing stuff end!
        hipEventRecord(stop, 0); // instrument code to measure end time
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time_ms, start, stop );

        printf("Back from kernel\n");
        hipMemcpy(colors, gpuColors, size_data, hipMemcpyDeviceToHost);

        printf("Frame %d/%d took %f ms.\n", frame+1, N_FRAMES, elapsed_time_ms); 

        for (int i = 0; i < IMG_WIDTH; ++i) {
            for (int j = IMG_HEIGHT - 1; j >= 0; --j) {

                int i2 = i, j2 = IMG_HEIGHT - j - 1;

                // float alphay = y / (float) (IMG_WIDTH - 1);
                // float alphaz = z / (float) (IMG_HEIGHT - 1);

                // Vec3 rel_viewpoint_pos = Vec3(focal_length, -0.5 + alphaz, 0.5 - alphay);
                // rel_viewpoint_pos = rot.RotateVector(rel_viewpoint_pos);

                // Vec3 direction = rel_viewpoint_pos.normalize();
            
                // Vec3 color = RayTrace(w, cameraPos, direction);

                // int ir = static_cast<int>(255.999 * color.x);
                // int ig = static_cast<int>(255.999 * color.y);
                // int ib = static_cast<int>(255.999 * color.z);
                Int3 i = colors[i2 * IMG_WIDTH + j2]; 


                imgout << i.x << ' ' << i.y << ' ' << i.z << '\n';

            }
            
        }

        imgout.close();
    }

    // for (SceneObject* s : w->objects)
    for (int i = 0; i < MAX_OBJS; ++i)
        delete w->objects[i];

    

    for (int i = 0; i < MAX_LGHT; ++i)
        delete w->lights[i];

    return 0;
}