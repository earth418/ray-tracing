#include "hip/hip_runtime.h"
#include "utils.h"

// #define min(x, y) (x) > (y) ? (y) : (x)

CUDA_HEAD float Q_rsqrt( float number )
{
	long i;
	float x2, y;
	const float threehalfs = 1.5F;

	x2 = number * 0.5F;
	y  = number;
	i  = * ( long * ) &y;                       // evil floating point bit level hacking
	i  = 0x5f3759df - ( i >> 1 );               // what the fuck? 
	y  = * ( float * ) &i;
	y  = y * ( threehalfs - ( x2 * y * y ) );   // 1st iteration
//	y  = y * ( threehalfs - ( x2 * y * y ) );   // 2nd iteration, this can be removed

	return y;
}

    // Appends together as column vectors
CUDA_HEAD Mat3x3::Mat3x3(Vec3 c0, Vec3 c1, Vec3 c2) {
    for (int i = 0; i < 3; ++i) {
        arr[i][0] = c0[i];
        arr[i][1] = c1[i];
        arr[i][2] = c2[i];
    }
}

CUDA_HEAD float Mat3x3::det() const {
    return arr[0][0] * (arr[1][1] * arr[2][2] - arr[2][1] * arr[1][2])
            - arr[0][1] * (arr[1][0] * arr[2][2] - arr[2][0] * arr[1][2])
            + arr[0][2] * (arr[1][0] * arr[2][1] - arr[1][1] * arr[2][0]); 
}

Vec3 Vec3::cross(Vec3 other) const {
    return Vec3(y * other.z - z * other.y,
                -(x * other.z - z * other.x),
                x * other.y - y * other.x);
}

CUDA_HEAD Vec3 operator*(float left, Vec3 right) {
    return right * left;
}

CUDA_HEAD Vec3 operator+(float left, Vec3 right) {
    return right + left;
}

CUDA_HEAD Quat::Quat(Vec3 vec, float angle) {
    const float halfan = 0.5 * angle;
    float cos_a = cos(halfan);
    float sin_a = sin(halfan);

    w = cos_a;
    Vec3 ijk = vec * sin_a;
    x = ijk.x, y = ijk.y, z = ijk.z;
}

CUDA_HEAD Quat::Quat(Vec3 vec) {
    w = 0.0, x = vec.x, y = vec.y, z = vec.z;
}

// Computes the Hamiltonian 
// product of two Quats
CUDA_HEAD Quat Quat::operator*(Quat o) const {
    Quat quat = Quat();
    quat.w = w * o.w - x * o.x - y * o.y - z * o.z;
    quat.x = w * o.x + x * o.w + y * o.z - z * o.y;
    quat.y = w * o.y - x * o.z + y * o.w + z * o.x;
    quat.z = w * o.z + x * o.y - y * o.x + z * o.w;
    return quat;
}


CUDA_HEAD Vec3 Quat::RotateVector(Vec3 vec) const {
    Quat prime = Quat(w, -x, -y, -z);
    Quat v = Quat(vec);
    Quat result = *this * v * prime;

    // Rotation * quat-Vector * Rot.inv
    return Vec3(result.x, result.y, result.z);
}
